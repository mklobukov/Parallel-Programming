#include "hip/hip_runtime.h"
/* Vector-matrix multiplication: Y = A * X.
 * Host code.
 * Author: Naga Kandasamy
 * Modified by Greg Matthews and Mark Klobukov
 * for CUDA assignment # 1 for ECEC 622
 * Date: 2/21/2017
*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include "vec_mat_mult_kernel.cu"

#define MIN_NUMBER 1
#define MAX_NUMBER 4


extern "C" void compute_gold(float*, const float*, const float*, unsigned int, unsigned int);
Matrix allocate_matrix_on_gpu(const Matrix);
Matrix allocate_matrix(int, int, int);
void copy_matrix_to_device(Matrix, const Matrix);
void copy_matrix_from_device(Matrix, const Matrix);
void vec_mat_mult_on_device_using_global_memory(const Matrix, const Matrix, Matrix);
void vec_mat_mult_on_device_using_shared_memory(const Matrix, const Matrix, Matrix);
void print_matrix(const Matrix);
void FreeDeviceMatrix(Matrix *);
void FreeMatrix(Matrix *);
void checkCUDAError(const char*);
float get_random_number(int, int);
int checkResults(float *, float *, int, float);


int 
main(int argc, char** argv) {
	// Matrices for the program
	Matrix  A; // N x N matrix
	Matrix  X; // N x 1 vector
	Matrix  Y_cpu, Y_gpu_1, Y_gpu_2; // N x 1 vector
	struct timeval start, stop;
	
	// Initialize the random number generator with a seed value 
	srand(time(NULL));
	
	// Check command line arguments
	if(argc > 1){
		printf("Error. This program accepts no arguments. \n");
		exit(0);
	}		
	 
	// Allocate and initialize the matrices
	A  = allocate_matrix(MATRIX_SIZE, MATRIX_SIZE, 1); // Create a random N x N matrix
	X  = allocate_matrix(MATRIX_SIZE, 1, 1); // Create a random N x 1 vector 
	Y_cpu  = allocate_matrix(MATRIX_SIZE, 1, 0); // Allocate memory for the output vectors
	Y_gpu_1 = allocate_matrix(MATRIX_SIZE, 1, 0); 
    Y_gpu_2 = allocate_matrix(MATRIX_SIZE, 1, 0);
 
    // compute the vector-matrix multiplication on the CPU for comparison    	
    gettimeofday(&start, NULL);
	compute_gold(Y_cpu.elements, A.elements, X.elements, A.num_rows, A.num_columns);
		gettimeofday(&stop, NULL);
		printf("Serial run time = %0.8f s. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));
	
	// Perform the vector-matrix multiplication on the GPU using global memory
    // Return the results in Y_gpu_1
	vec_mat_mult_on_device_using_global_memory(A, X, Y_gpu_1);

	
   
	// check if the device result is equivalent to the expected solution
    printf("Checking against reference result. \n");
	int size_elements = NUM_ROWS;
	int res = checkResults(Y_cpu.elements, Y_gpu_1.elements, size_elements, 0.0001);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");


    // Perform the vector-matrix multiplication on the GPU using shared memory
    // Return the results in Y_gpu_2
	vec_mat_mult_on_device_using_shared_memory(A, X, Y_gpu_2);
   
	//print_matrix(Y_cpu);
	//printf("\n");
	//print_matrix(Y_gpu_1);	
	
	

	// check if the device result is equivalent to the expected solution
    printf("Checking against reference result. \n");
    res = checkResults(Y_cpu.elements, Y_gpu_2.elements, size_elements, 0.0001);
	printf("Test %s\n", (1 == res) ? "PASSED" : "FAILED");
	/*
	printf("REF MATRIX: \n");
	print_matrix(Y_cpu);
	printf("SHARED PROGRAM RESULT: \n");
	print_matrix(Y_gpu_2);
	*/
	// Free host matrices
	free(A.elements); A.elements = NULL;
	free(X.elements); X.elements = NULL;
	free(Y_cpu.elements); Y_cpu.elements = NULL;
	free(Y_gpu_1.elements); Y_gpu_1.elements = NULL;
    free(Y_gpu_2.elements); Y_gpu_2.elements = NULL;

	return 0;
}

// Complete the functionality of vector-matrix multiplication using the GPU 
// Kernel should use global memory
void 
vec_mat_mult_on_device_using_global_memory(const Matrix A, const Matrix X, Matrix Y)
{
	struct timeval start, stop;

	Matrix d_A = allocate_matrix_on_gpu(A);
	Matrix d_X = allocate_matrix_on_gpu(X);
	Matrix d_Y = allocate_matrix_on_gpu(Y);
	
	copy_matrix_to_device(d_A, A);
	copy_matrix_to_device(d_X, X);
	
	dim3 threads(512, 1);
	dim3 grid(MATRIX_SIZE/ threads.x, 1);

	gettimeofday(&start, NULL);	
	/* Execute the kernel. */
	vec_mat_kernel_naive<<< grid, threads >>>(d_A.elements, d_X.elements, d_Y.elements);
	hipDeviceSynchronize();

    gettimeofday(&stop, NULL);
	printf("Execution time = %fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));

    checkCUDAError("Error in kernel");/* Check if execution generated an error. */

	copy_matrix_from_device(Y, d_Y);  /* Read Y from the device. */
	
    FreeDeviceMatrix(&d_A);			  /* Free device matrices. */
	FreeDeviceMatrix(&d_X);
	FreeDeviceMatrix(&d_Y);
}

// Complete the functionality of vector-matrix multiplication using the GPU
// Kernel should use shared memory
void 
vec_mat_mult_on_device_using_shared_memory(const Matrix A, const Matrix X, Matrix Y)
{
struct timeval start, stop;

	Matrix d_A = allocate_matrix_on_gpu(A);
	Matrix d_X = allocate_matrix_on_gpu(X);
	Matrix d_Y = allocate_matrix_on_gpu(Y);
	
	copy_matrix_to_device(d_A, A);
	copy_matrix_to_device(d_X, X);
	
	dim3 threads(BLOCK_DIM_X, 1);
	dim3 grid(MATRIX_SIZE/ threads.x, 1);

	gettimeofday(&start, NULL);	
	/* Execute the kernel. */
	vec_mat_kernel_optimized<<< grid, threads >>>(d_A.elements, d_X.elements, d_Y.elements);
	hipDeviceSynchronize();

    gettimeofday(&stop, NULL);
	printf("Execution time = %.5fs. \n", (float)(stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec)/(float)1000000));

    checkCUDAError("Error in kernel");/* Check if execution generated an error. */

	copy_matrix_from_device(Y, d_Y);  /* Read Y from the device. */	
    FreeDeviceMatrix(&d_A);			  /* Free device matrices. */
	FreeDeviceMatrix(&d_X);
	FreeDeviceMatrix(&d_Y);
}


// Allocate a device matrix of same size as M.
Matrix 
allocate_matrix_on_gpu(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.num_rows * M.num_columns * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix 
allocate_matrix(int num_rows, int num_columns, int init)
{
    	Matrix M;
    	M.num_columns = M.pitch = num_columns;
    	M.num_rows = num_rows;
    	int size = M.num_rows * M.num_columns;
		
	M.elements = (float*) malloc(size*sizeof(float));
	for(unsigned int i = 0; i < size; i++){
		if(init == 0) M.elements[i] = 0; 
		else
			M.elements[i] = get_random_number(MIN_NUMBER, MAX_NUMBER);
	}
    return M;
}	

// Copy a host matrix to a device matrix.
void 
copy_matrix_to_device(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.num_rows * Mhost.num_columns * sizeof(float);
    Mdevice.num_rows = Mhost.num_rows;
    Mdevice.num_columns = Mhost.num_columns;
    Mdevice.pitch = Mhost.pitch;
    hipMemcpy(Mdevice.elements, Mhost.elements, size, hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void 
copy_matrix_from_device(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.num_rows * Mdevice.num_columns * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, hipMemcpyDeviceToHost);
}

// Prints the matrix out to screen
void 
print_matrix(const Matrix M)
{
	for(unsigned int i = 0; i < M.num_rows; i++){
		for(unsigned int j = 0; j < M.num_columns; j++)
			printf("%f ", (float)M.elements[i*M.num_columns + j]);
		printf("\n");
	} 
	printf("\n");
}

// Returns a random floating-point number between the specified min and max values 
float 
get_random_number(int min, int max){
	return (float)floor((double)(min + (max - min + 1)*((float)rand()/(float)RAND_MAX)));
}

int 
checkResults(float *reference, float *gpu_result, int num_elements, float threshold)
{
    int checkMark = 1;
    float epsilon = 0.0;
    
    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > threshold){
            checkMark = 0;
            break;
        }

    for(int i = 0; i < num_elements; i++)
        if(fabsf((reference[i] - gpu_result[i])/reference[i]) > epsilon){
            epsilon = fabsf((reference[i] - gpu_result[i])/reference[i]);
        }

    printf("Max epsilon = %f. \n", epsilon); 
    return checkMark;
}

void 
FreeDeviceMatrix(Matrix* M)                                 /* Free a device matrix. */
{
	hipFree(M->elements);
	M->elements = NULL;
}

// Free a host Matrix
void 
FreeMatrix(Matrix* M)
{
	free(M->elements);
	M->elements = NULL;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		printf("CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}

